#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>
#include <helper_timer.h>

hipError_t cublasMultiply(float *a, float *b, float *c, unsigned int na,unsigned int nb);

int main()
{
	float a[]={ 0.1,0.2,0.3,0.4};

	float b[]={ 1.1,1.2,1.3,
			    2.1,2.2,2.3,
				3.1,3.2,3.3,
				4.1,4.2,4.3};

	float* c = new float[3];
	//memset(c,0,sizeof(float));
	for(int i=0;i<3;i++)
		c[i]=i;

	int colsa=4;
	int rowsa=1;

	int colsb=3;
	int rowsb=4;

	float* dptra;
	float* dptrb;
	float* dptrc;

	hipMalloc((void**)&dptra,4*sizeof(float));
	hipMalloc((void**)&dptrb,12*sizeof(float));
	hipMalloc((void**)&dptrc,3*sizeof(float));

	hipMemcpy(dptra,a,4*sizeof(float),hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(dptrb,b,12*sizeof(float),hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(dptrc,c,3*sizeof(float),hipMemcpyKind::hipMemcpyHostToDevice);


    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // Do the actual multiplication
	float alpha=1;
	float beta=0;
	/*
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
	*/
	hipblasStatus_t res = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 1,3, 4, &alpha, dptra, 1, dptrb, 3, &beta, dptrc, 1);
	//hipblasStatus_t res = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1,4, 3, &alpha, dptra, 1, dptrb, 4, &beta, dptrc, 1);
    // Destroy the handle
    hipblasDestroy(handle);
	
	printf("%d\n",res);

	hipMemcpy(c,dptrc,3*sizeof(float),hipMemcpyKind::hipMemcpyDeviceToHost);

	for(int i=0;i<3;i++)
	{
		printf("%f ",c[i]);
	}


	delete[] c;
    hipDeviceReset();
    return 0;
}
void gpu_blas_mmul(const float *A, const float *B, float *C, unsigned int m, unsigned int k, unsigned int n) {
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;


    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // Do the actual multiplication
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
    // Destroy the handle
    hipblasDestroy(handle);
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t cublasMultiply(float *a, float *b, float *c, unsigned int na,unsigned int nb)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;


    cudaStatus = hipSetDevice(0);
    cudaStatus = hipMalloc((void**)&dev_c, nb * sizeof(float));
    cudaStatus = hipMalloc((void**)&dev_a, na * sizeof(float));
    cudaStatus = hipMalloc((void**)&dev_b, na*nb * sizeof(float));

    cudaStatus = hipMemcpy(dev_a, a, na * sizeof(float), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_b, b, nb*na * sizeof(float), hipMemcpyHostToDevice);

	//gpu_blas_mmul(dev_a, dev_b, dev_c, nr_rows_A, nr_cols_A, nr_cols_B);
	gpu_blas_mmul(dev_a, dev_b, dev_c, 1, na, nb);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(c, dev_c, nb * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
