#include "TsCudaANNFunctions.cuh"
#include <Windows.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


void printDeviceData(float* dptr,int n)
{
	float* res = new float[n];
	
	hipMemcpy(res,dptr,n*sizeof(float),hipMemcpyKind::hipMemcpyDeviceToHost);

	printf("\n");
	for(int i=0;i<n;i++)
		printf("%f\t",res[i]);
	printf("\n");

	delete[] res;
}
void printDeviceDataD(double* dptr,int n)
{
	double* res = new double[n];
	
	hipMemcpy(res,dptr,n*sizeof(double),hipMemcpyKind::hipMemcpyDeviceToHost);

	printf("\n");
	for(int i=0;i<n;i++)
		printf("%lf\t",res[i]);
	printf("\n");

	delete[] res;
}
void printDeviceDataD(int* dptr,int n)
{
	int* res = new int[n];
	
	hipMemcpy(res,dptr,n*sizeof(int),hipMemcpyKind::hipMemcpyDeviceToHost);

	printf("\n");
	for(int i=0;i<n;i++)
		printf("%d\t",res[i]);
	printf("\n");

	delete[] res;
}
int main()
{
    int layers = 4;
	int* layerssize = new int[4];
	
	/*
	layerssize[0]=2;
	layerssize[1]=1024;
	layerssize[2]=2048;
	layerssize[3]=32;
	layerssize[4]=1;
	*/

	layerssize[0]=2048*0+2;
	layerssize[1]=2048*2*0+512;
	layerssize[2]=256;
	layerssize[3]=1;

	Network* thenetwork;
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	StopWatchInterface *timer = 0;
	sdkCreateTimer(&timer);


	int size = 0;

	for(int i=0;i<layers;i++)
	{
		size+=layerssize[i]*5;
		if(i<layers-1)
			size+=layerssize[i]*layerssize[i+1];
	}
	size*=sizeof(float);


	printf("total %f mb\n",size/1024.0f/1024.0f);


	hipError_t cudaStatus = hostStartNetwork(layerssize,layers,thenetwork);

	printDeviceDataD(thenetwork->device->sumsW,thenetwork->host->layers-1);

	//getchar();

    if (cudaStatus != hipSuccess) {fprintf(stderr, "addWithCuda failed!");return 1;}

	float* inputs = new float[layerssize[0]];
	float* results = new float[1];
	float* expected = new float[1];

	for(int i=0;i<layerssize[0];i++)
		inputs[i]=0;

	*expected=0;

	inputs[0]=1;
	inputs[1]=0;
	
	sdkStartTimer(&timer);

	for(int i=0;i<100000;i++)
	{
		int x = i%4;
		inputs[0]=x/2;
		inputs[1]=x%2;
		*expected=(x==1||x==2);
		hostCudaTrain(thenetwork,handle,inputs,0.5,0.05,expected,NULL);
	}

	sdkStopTimer(&timer);
	float reduceTime = sdkGetAverageTimerValue(&timer);

	printf("Time:%f\n",reduceTime);
	
	for(int i=0;i<4;i++)
	{
		inputs[0]=i/2;
		inputs[1]=i%2;
		*results=0;

		hostCudaForward(thenetwork,handle,inputs,0.5,results,NULL);

		printf("(%f,%f)=>%f\n",inputs[0],inputs[1],results[0]);
	}	
	
	disposeNetwork(thenetwork);


	cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hipDeviceReset failed!");return 1;}
    return 0;
}
