#include "../../TsCudaANN/TsCudaANN/TsCudaANNFunctions.cuh"

struct TheData
{
	Network* thenetwork;
	hipblasHandle_t cublash;
};

TheData* WINAPI cudaANNCreateNetwork(int* layerssize,int layers)
{
	Network* thenetwork;
	hipblasHandle_t cublash;

	if(hostStartNetwork(layerssize,layers,thenetwork)!=hipError_t::hipSuccess) return NULL;
	hipblasCreate(&cublash);

	TheData* thedata = new TheData();
	thedata->thenetwork=thenetwork;
	thedata->cublash=cublash;

	return thedata;
}

bool WINAPI cudaANNTrain(TheData* thedata,float* inputs,float* expected,float alpha,float learningrate,float &totalerror) 
{
	int outputsize = thedata->thenetwork->host->sizesN[thedata->thenetwork->host->layers-1];
	float* errors = new float[outputsize];
	if(hostCudaTrain(thedata->thenetwork,thedata->cublash,inputs,alpha,learningrate,expected,errors)!=hipError_t::hipSuccess)
		return false;
	totalerror=0;
	for(int i=0;i<outputsize;i++)
		totalerror+=errors[i]*errors[i];
	totalerror*=0.5;
	return true;
}

bool WINAPI cudaANNForward(TheData* thedata,float* inputs,float* outputs,float alpha)
{
	return hostCudaForward(thedata->thenetwork,thedata->cublash,inputs,alpha,outputs,NULL)==hipError_t::hipSuccess;
}

bool WINAPI cudaANNFree(TheData* thedata)
{
	hipblasDestroy(thedata->cublash);
	hipHostFree(thedata->thenetwork);
	delete thedata;
	return true;
}

bool WINAPI cudaANNBackup(TheData* thedata,char* &ptr,int &size)
{
	Network* thenetwork = thedata->thenetwork;
	int totalw = thenetwork->host->totalw;
	int totaln = thenetwork->host->totaln;	
	int layers = thenetwork->host->layers;

	size = (1 + layers + totalw + totaln)*sizeof(int) + (totaln + totalw)*sizeof(float);
	
	ptr = (char*)malloc(size);
	char* iptr = ptr;

	*((int*)iptr) = layers; iptr+=sizeof(int);

	for (int i = 0; i < layers; i++)
	{
		*((int*)iptr) = thenetwork->host->sizesN[i];
		iptr += sizeof(int);
	}

	*((int*)iptr) = totaln; iptr += sizeof(int);
	hipMemcpy(iptr, thenetwork->host->bias[0], totaln*sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);
	iptr += totaln*sizeof(float);

	*((int*)iptr) = totalw; iptr += sizeof(int);
	hipMemcpy(iptr, thenetwork->host->weights[0], totalw*sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);
	iptr += totalw*sizeof(float);

	return true;
}

bool WINAPI cudaANNRestore(TheData* thedata, char* weights, char* bias)
{
	int totalw=thedata->thenetwork->host->totalw;
	int totaln=thedata->thenetwork->host->totaln;

	hipMemcpy(thedata->thenetwork->host->bias[0], bias, totaln*sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(thedata->thenetwork->host->weights[0], weights, totalw*sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);

	return true;
}