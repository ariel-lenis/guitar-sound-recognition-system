#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hipfft/hipfft.h>
#include "windows.h"
#include <hip/device_functions.h>

#define MAX_THREADS_PER_BLOCK 512
#define MAX_BLOCKS_PER_DIMENSION 65535

//calculate the optimal number of threads and blocks two of then on 1 dimension
void blocksAndThreads(int n,int maxthreads,int &blocks,int &threads)
{	
	blocks = (n+MAX_THREADS_PER_BLOCK-1)/MAX_THREADS_PER_BLOCK;
	threads= (n<MAX_THREADS_PER_BLOCK)?n:MAX_THREADS_PER_BLOCK;
}
//the limit of blocks per dimension is 2^16-1, so we need to use a 2d block
void blocksAndThreads(int n,int maxthreads,dim3 &blocks,int &threads)
{	
	int _blocks = (n+MAX_THREADS_PER_BLOCK-1)/MAX_THREADS_PER_BLOCK;
	threads= (n<MAX_THREADS_PER_BLOCK)?n:MAX_THREADS_PER_BLOCK;

	int _blocks2=1;
	if(_blocks>MAX_BLOCKS_PER_DIMENSION)
	{
		_blocks2=(_blocks+MAX_BLOCKS_PER_DIMENSION-1)/MAX_BLOCKS_PER_DIMENSION;
		_blocks=MAX_BLOCKS_PER_DIMENSION;
	}
	blocks=dim3(_blocks,_blocks2);
}

bool _stdcall Test()
{
	int count=0;
	hipGetDeviceCount(&count);
	return count>0;
}
void _stdcall FFT(void* input,void* output,int n,int direction)
{
	int* cudaData = NULL;
	hipfftHandle plan;

	hipSetDevice(0);
	hipMalloc((void**)&cudaData, 2 * n * sizeof(float));
	hipMemcpy(cudaData, input, 2*n*sizeof(float), hipMemcpyHostToDevice);
	hipfftPlan1d(&plan,n,HIPFFT_C2C,1);

	if(direction==1)
		hipfftExecC2C(plan,(hipfftComplex*)cudaData,(hipfftComplex*)cudaData,HIPFFT_FORWARD);
	else
		hipfftExecC2C(plan,(hipfftComplex*)cudaData,(hipfftComplex*)cudaData,HIPFFT_BACKWARD);

	hipfftDestroy(plan);
	hipMemcpy(output, cudaData, 2*n*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(cudaData);
}

void _stdcall MultipleFFT(void* input,void* output,int blocksize,int nblocks,int direction)
{
	int* cudaData = NULL;
	hipfftHandle plan;

	int allsize = blocksize*nblocks;

	hipSetDevice(0);
	hipMalloc((void**)&cudaData, 2 * allsize * sizeof(float));
	hipMemcpy(cudaData, input, 2*allsize*sizeof(float), hipMemcpyHostToDevice);
	hipfftPlan1d(&plan,blocksize,HIPFFT_C2C,nblocks);

	if(direction==1)
		hipfftExecC2C(plan,(hipfftComplex*)cudaData,(hipfftComplex*)cudaData,HIPFFT_FORWARD);
	else
		hipfftExecC2C(plan,(hipfftComplex*)cudaData,(hipfftComplex*)cudaData,HIPFFT_BACKWARD);

	hipfftDestroy(plan);
	hipMemcpy(output, cudaData, 2*allsize*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(cudaData);
}

/*
__global__ void startWaveA(float* ptr,int n,int target)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;	
	if(i>=target) return;
	if(i<n/2)
	{
		ptr[2*(n-1-i)]=ptr[n-1-i];
		ptr[n-1-i]=0;
	}
	ptr[2*(n-i)-1]=ptr[i];	
	ptr[i]=0;
}
__global__ void startWaveB(float* ptr,int n,int target)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;	
	if(i>=target) return;
	ptr[i*2]=ptr[2*(n-i)-1];
	ptr[2*(n-i)-1]=0;
}

float* ptr = NULL;
int n;
float** sptr = NULL;
int srows;

bool _stdcall SetWave(float* data,int _n)
{
	if(ptr!=NULL) hipFree(ptr);
	n=_n;

	float* ptr=NULL;
	if(hipSetDevice(0)!=hipSuccess) return false;
	if(hipMalloc((void**)&ptr,2*n*sizeof(float))!=hipSuccess) return false;

	if(hipMemcpy(ptr, data, n*sizeof(float), hipMemcpyHostToDevice)!=hipSuccess)
	{
		hipFree(ptr);
		return false;
	}

	int target = (n+1)/2;
	int blocks = (target+MAX_THREADS_PER_BLOCK-1)/MAX_THREADS_PER_BLOCK;

	startWaveA<<<blocks,MAX_THREADS_PER_BLOCK>>>(ptr,n,target);
	startWaveB<<<blocks,MAX_THREADS_PER_BLOCK>>>(ptr,n,target);
	//hipFree(ptr);

	return true;
}
*/
/*
float* ptr = NULL;
int n;
float** sptr = NULL;
int srows;
float* buffer;
*/


float* _stdcall HostCloneWaveToDevice(float* data,int n)
{
	float *ptr;
	if(hipSetDevice(0)!=hipSuccess) return NULL;
	if(hipMalloc((void**)&ptr,n*sizeof(float))!=hipSuccess) return false;

	if(hipMemcpy(ptr, data, n*sizeof(float), hipMemcpyHostToDevice)!=hipSuccess)
	{
		hipFree(ptr);
		return NULL;
	}

	return ptr;
}

bool _stdcall HostFastFourierTransform(float* deviceData,int n,int direction)
{
	//int* cudaData = NULL;
	hipfftHandle plan;
	hipSetDevice(0);

	hipfftPlan1d(&plan,n,HIPFFT_C2C,1);

	if(direction==1)
	{
		if(hipfftExecC2C(plan,(hipfftComplex*)deviceData,(hipfftComplex*)deviceData,HIPFFT_FORWARD)!=hipfftResult_t::HIPFFT_SUCCESS)
			goto error;
	}
	else
	{
		if(hipfftExecC2C(plan,(hipfftComplex*)deviceData,(hipfftComplex*)deviceData,HIPFFT_BACKWARD)!=hipfftResult_t::HIPFFT_SUCCESS)
			goto error;
	}

	hipfftDestroy(plan);
	return true;
error:
	hipfftDestroy(plan);
	//hipFree(cudaData);	
}

__global__ void deviceMultiply(float* where,float* a,float* b,unsigned int n)
{
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx>=n) return;
	//where[idx]=a[idx]*b[idx];

	float va = a[2*idx];
	float vb = a[2*idx+1];

	float vc = b[2*idx];
	float vd = b[2*idx+1];


	where[2*idx]=va*vc-vb*vd;
	where[2*idx+1]=va*vd+vb*vc;

}
__device__ const float pi = (float)3.14159265358979323846;

__device__ const double multiplier = 1.8827925275534296252520792527491;
__device__ float FTWavelet( float x, float scale, float f0 )
{
    if ( x < 0.9 / scale ||  x > 1.1 / scale ) {
        return (float)0.0;
    }

	double two_pi_f0 = 2.0 * pi * f0;


    scale *= (float)f0;

    // 1.88279*exp(-0.5*(2*pi*x*10-2*pi*10)^2)

    float basic = (float)(multiplier *
            exp(-0.5*(2*pi*x*scale-two_pi_f0)*(2*pi*x*scale-two_pi_f0)));

    // pi^0.25*sqrt(2.0)*exp(-0.5*(2*pi*x*scale-2*pi*0.849)^2)
    return sqrt(scale)*basic;
}


__global__ void deviceModule(float* where,float* from,unsigned int n)
{
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx>=n) return;
	float dx = from[idx*2];
	float dy = from[idx*2+1];
	where[idx]=log(sqrt(dx*dx+dy*dy));
}

bool _stdcall HostSemiConvolution(hipfftHandle cuplan,float* fftdeviceA,float* fftdeviceB,float* devicebuffer,float* deviceresult,unsigned int n)
{
	int blocks,threads;
	hipError_t cudaStatus;
	blocksAndThreads(n,MAX_THREADS_PER_BLOCK,blocks,threads);
	deviceMultiply<<<blocks,threads>>>(devicebuffer,fftdeviceA,fftdeviceB,n);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) goto error;
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) goto error;

	if(hipfftExecC2C(cuplan,(hipfftComplex*)devicebuffer,(hipfftComplex*)devicebuffer,HIPFFT_BACKWARD)!=hipfftResult_t::HIPFFT_SUCCESS)
		goto error;
	//if(!HostFastFourierTransform(devicebuffer,n,-1)) goto error;

	blocksAndThreads(n,MAX_THREADS_PER_BLOCK,blocks,threads);
	deviceModule<<<blocks,threads>>>(deviceresult,devicebuffer,n);	
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) goto error;
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) goto error;

	return true;
error:
	return false;
}
__global__ void deviceRealToComplex(float* where,float* from,unsigned int n)
{
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx>=n) return;
	where[2*idx]=from[idx];
	where[2*idx+1]=0;
}
__global__ void deviceZeroMemory(float* where,unsigned int n)
{
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx>=n) return;
	where[idx]=0;
}

struct ScalogramPlan
{
	float* fftdevicea;
	float* fftdeviceb;
	float* fftdevicebuffer;
	float* fftdeviceresult;	
	hipfftHandle cuplan;
	int n;
};

ScalogramPlan* WINAPI HostPlanScalogram(float* deviceData,int n)
{
	ScalogramPlan* plan = new ScalogramPlan();
	hipError_t cudaStatus;

	float* fftdevicea=NULL;
	float* fftdeviceb=NULL;
	float* fftdevicebuffer=NULL;
	float* fftdeviceresult=NULL;
	hipfftHandle cuplan=NULL;

	if(hipMalloc((void**)&fftdevicea,2*n*sizeof(float))!=hipSuccess) goto error;		
	if(hipMalloc((void**)&fftdeviceb,2*n*sizeof(float))!=hipSuccess) goto error;
	if(hipMalloc((void**)&fftdevicebuffer,2*n*sizeof(float))!=hipSuccess) goto error;
	if(hipMalloc((void**)&fftdeviceresult,n*sizeof(float))!=hipSuccess) goto error;

	int blocks,threads;
	blocksAndThreads(n,MAX_THREADS_PER_BLOCK,blocks,threads);
	deviceRealToComplex<<<blocks,threads>>>(fftdevicea,deviceData,n);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) goto error;

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) goto error;

	blocksAndThreads(2*n,MAX_THREADS_PER_BLOCK,blocks,threads);
	deviceZeroMemory<<<blocks,threads>>>(fftdeviceb,2*n);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) goto error;

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) goto error;	

	hipfftPlan1d(&cuplan,n,HIPFFT_C2C,1);

	hipfftPlan1d(&cuplan,n,HIPFFT_C2C,1);
	if(hipfftExecC2C(cuplan,(hipfftComplex*)fftdevicea,(hipfftComplex*)fftdevicea,HIPFFT_FORWARD)!=hipfftResult_t::HIPFFT_SUCCESS) goto error;

	plan->fftdevicea=fftdevicea;
	plan->fftdeviceb=fftdeviceb;
	plan->fftdevicebuffer=fftdevicebuffer;
	plan->fftdeviceresult=fftdeviceresult;
	plan->cuplan=cuplan;
	plan->n=n;


	return plan;
error:
	MessageBoxA(NULL,hipGetErrorString(cudaStatus),NULL,0);
	if(fftdevicea!=NULL)		hipFree(fftdevicea);
	if(fftdeviceb!=NULL)		hipFree(fftdeviceb);
	if(fftdevicebuffer!=NULL)	hipFree(fftdevicebuffer);
	if(fftdeviceresult!=NULL)	hipFree(fftdeviceresult);
	return NULL;
}


__device__ float Morlet(int i,int n)
{
    float range = 4;
    float x=-range+2.0f*range*i/n;
	float amplitude=1;
    return (float)(amplitude * (exp(-x*x/2)*cos(5*x)));
}

__global__ void devicePrepareWindow(float* where,unsigned int n,unsigned int wn)
{
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx>=n) return;
	
	if(idx<wn)	where[2*idx]= Morlet(idx,wn);//window[idx];
	else		where[2*idx]=0;
	where[2*idx+1]=0;
}

/*
__global__ void devicePrepareWindow(float* where,float* window,unsigned int n,unsigned int wn)
{
	int idx = blockDim.x*blockIdx.x+threadIdx.x;
	if(idx>=n) return;
	if(idx>n/2)
	{
		where[2*idx]=0;
		where[2*idx+1]=0;	
	}
	else
	{
		float f0=82;
		where[2*idx]=FTWavelet(idx, (float)0.1*wn/n, f0 );
		where[2*idx+1]=0;
	}
}
*/
int WINAPI HostIterateWindow(int wn,ScalogramPlan* plan,float* hostresult)
{
	int blocks,threads;
	hipError_t cudaStatus=hipError_t::hipSuccess;
	float* deviceWindow;
	
	hipGetLastError();
	
	//hipMalloc((void**)&deviceWindow,wn*sizeof(float));
	//hipMemcpy(deviceWindow,window,wn*sizeof(float),hipMemcpyKind::hipMemcpyHostToDevice);

	blocksAndThreads(plan->n,MAX_THREADS_PER_BLOCK,blocks,threads);
	devicePrepareWindow<<<blocks,threads>>>(plan->fftdeviceb,plan->n,wn);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) goto error;
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) goto error;

	//hipFree(deviceWindow);


	if(hipfftExecC2C(plan->cuplan,(hipfftComplex*)plan->fftdeviceb,(hipfftComplex*)plan->fftdeviceb,HIPFFT_FORWARD)!=hipfftResult_t::HIPFFT_SUCCESS)
		goto error;

	//if(!HostFastFourierTransform(plan->fftdeviceb,plan->n,1)) goto error;


	if(!HostSemiConvolution(plan->cuplan,plan->fftdevicea,plan->fftdeviceb,plan->fftdevicebuffer,plan->fftdeviceresult,plan->n)) goto error;
	
	if(hipMemcpy(hostresult,plan->fftdeviceresult,plan->n*sizeof(float),hipMemcpyKind::hipMemcpyDeviceToHost)!=hipError_t::hipSuccess)
		goto error;
	return 1;

error:
	return 0;
}


void WINAPI HostDestroyScalogram(ScalogramPlan* data)
{
	hipFree(data->fftdevicea);
	hipFree(data->fftdeviceb);
	hipFree(data->fftdevicebuffer);
	hipFree(data->fftdeviceresult);
	hipfftDestroy(data->cuplan);
}



__global__ void	prepareSpectrogram(float* data,int datasize,float* buffer,int buffersize,float* cuwindow,int fftsize,int samplesrequired,float p)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=buffersize) return;
	int peakpos = id/fftsize;
	int epos =id%fftsize;

	if((peakpos==0 && epos<=fftsize/2) || (peakpos==samplesrequired-1 && epos>=fftsize/2))
	{
		buffer[id*2] = 0;
		buffer[id*2+1]=0;		
		return;
	}

	int pos = (int)(peakpos*p)-fftsize/2+epos;

	if(pos<0||pos>=datasize) return;
	buffer[id*2] = data[pos]*cuwindow[epos];
	buffer[id*2+1]=0;
}
__global__ void complexToReal(float* buffer,float* buffermodule,int finalsize,int fftsize)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=finalsize) return;

	int partsize = (fftsize+1)/2;
	int partid = id/partsize;
	int fftid = id%partsize;

	int pos = partid*fftsize*2+fftid*2;

	float dr = buffer[pos]*buffer[pos];
	float di = buffer[pos+1]*buffer[pos+1];
	buffermodule[id]=sqrtf(dr*dr+di*di);	
}
__global__ void cudaMean(float* xbuffer,unsigned int jump,unsigned int n)
{
	unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;

	if(id*jump*2<n)
	{
		//xbuffer[id*jump*2]=xbuffer[id*jump*2];
		if(id*jump*2+jump<n)
			xbuffer[id*jump*2]+=xbuffer[id*jump*2+jump];
		if(jump==1) xbuffer[id*jump*2]/=n;
	}
}
__global__ void cudaLog(float* xbuffer,unsigned int n)
{
	unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;

	if(id>=n) return;
	xbuffer[id]=logf(xbuffer[id]*10);
	xbuffer[id]+=10;
	if(xbuffer[id]<0) xbuffer[id]=0;
}
__global__ void cudaStd(float* xbuffer,float mean,unsigned int n)
{
	unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=n) return;
	float delta = xbuffer[id]-mean;
	xbuffer[id]=delta*delta;
}
__global__ void cudaNormalize(float* buffer,float mean,float std,int n)
{
	unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=n) return;

	float val = buffer[id];
	if(val>mean+std) val=mean+std;
	else if(val<mean-std) val=mean-std;

	//val=(val-mean)/std;

	//if(val<0) buffer[id]=0;

	float logk =1;
	if(std==0)	buffer[id]=0;
	else
		buffer[id] = (float)((logf(logk + val)-logf(logk)) / (logf(logk + mean+std)-logf(logk)));
}
//void STE()
float Mean(float* cuBuffer,int n,bool readonly)
{
	float* xbuffer=NULL;

	hipSetDevice(0);

	if(readonly)
	{
		if(hipMalloc((void**)&xbuffer,n*sizeof(float))!=hipSuccess) goto error;		
		if(hipMemcpy( xbuffer, cuBuffer, n*sizeof(float), hipMemcpyDeviceToDevice)!=hipSuccess) goto error;
	}
	else
		xbuffer=cuBuffer;
	
	int pow = 0,spow = 1;
	while(spow<n) {pow++;spow*=2;}

	int xn=n;
	unsigned int jump=1;
	for(int i=0;i<pow;i++)
	{
		xn=(xn+1)/2;
		int blocks = (xn+MAX_THREADS_PER_BLOCK-1)/MAX_THREADS_PER_BLOCK;
		cudaMean<<<blocks,MAX_THREADS_PER_BLOCK>>>(xbuffer,jump,(unsigned int)n);
		jump*=2;
	}

	float result = 0;
	if(hipMemcpy( &result, xbuffer, sizeof(float), hipMemcpyDeviceToHost)!=hipSuccess) goto error;

	if(readonly)
		hipFree(xbuffer);

	return result;
error:
	if(xbuffer!=NULL)	hipFree(xbuffer);
	return 0;
}

float StandardDeviation(float* cuBuffer,int n,float* _mean)
{
	float* xbuffer=NULL;

	hipSetDevice(0);	

	if(hipMalloc((void**)&xbuffer,n*sizeof(float))!=hipSuccess) goto error;		
	if(hipMemcpy( xbuffer, cuBuffer, n*sizeof(float), hipMemcpyDeviceToDevice)!=hipSuccess) goto error;

	int blocks;

	float mean = Mean(xbuffer,n,false);

	if(hipMemcpy( xbuffer, cuBuffer, n*sizeof(float), hipMemcpyDeviceToDevice)!=hipSuccess) goto error;

	blocks = (n+MAX_THREADS_PER_BLOCK-1)/MAX_THREADS_PER_BLOCK;
	cudaStd<<<blocks,MAX_THREADS_PER_BLOCK>>>(xbuffer,mean,(unsigned int)n);
	
	float std2 = Mean(xbuffer,n,false);
	*_mean=mean;

	hipFree(xbuffer);
	
	return sqrt(std2);
error:
	if(xbuffer!=NULL)	hipFree(xbuffer);
	return 0;
}


float* _stdcall Spectrogram(float* ptr,int n,int fftsize,int samplesrequired,float* window,float* output,float* mean,float* std)
{
	float* sptr=NULL;
	float p = (float)n/(samplesrequired-1);
	float* buffer=NULL;
	float* buffermodule=NULL;
	float* cuwindow=NULL;
	int buffersize = samplesrequired*fftsize;//but in complex the real size is 2*buffersize

	hipSetDevice(0);
	if(hipMalloc((void**)&buffer,2*buffersize*sizeof(float))!=hipSuccess) goto error;		
	if(hipMalloc((void**)&cuwindow,fftsize*sizeof(float))!=hipSuccess) goto error;
	if(hipMemcpy(cuwindow, window, fftsize*sizeof(float), hipMemcpyHostToDevice)!=hipSuccess) goto error;

	int blocks = (buffersize+MAX_THREADS_PER_BLOCK-1)/MAX_THREADS_PER_BLOCK;
	prepareSpectrogram<<<blocks,MAX_THREADS_PER_BLOCK>>>(ptr,n,buffer,buffersize,cuwindow,fftsize,samplesrequired,p);
	hipFree(cuwindow);

	hipfftHandle plan;

	int allsize = fftsize*samplesrequired;
	
	hipfftPlan1d(&plan,fftsize,HIPFFT_C2C,samplesrequired);
	if(hipfftExecC2C(plan,(hipfftComplex*)buffer,(hipfftComplex*)buffer,HIPFFT_FORWARD)!=hipfftResult_t::HIPFFT_SUCCESS) goto error;
	hipfftDestroy(plan);

	int finalsize = ((fftsize+1)/2)*samplesrequired;//aprox. just the half of the fft have the data because nquyst teorem
	if(hipMalloc((void**)&buffermodule,finalsize*sizeof(float))!=hipSuccess) goto error;	
	
	blocks = (finalsize+MAX_THREADS_PER_BLOCK-1)/MAX_THREADS_PER_BLOCK;
	complexToReal<<<blocks,MAX_THREADS_PER_BLOCK>>>(buffer,buffermodule,finalsize,fftsize);
	hipFree(buffer);

	//*mean = Mean(buffermodule,finalsize,true);

	*std = StandardDeviation(buffermodule,finalsize,mean);


	cudaNormalize<<<blocks,MAX_THREADS_PER_BLOCK>>>(buffermodule,*mean,*std,finalsize);


	hipMemcpy(output, buffermodule, finalsize*sizeof(float), hipMemcpyDeviceToHost);	
	hipFree(buffermodule);
	return sptr;
error:
	if(sptr!=NULL)	hipFree(sptr);
	if(buffer!=NULL)	hipFree(buffer);
	if(cuwindow!=NULL)	hipFree(cuwindow);
	return NULL;
}